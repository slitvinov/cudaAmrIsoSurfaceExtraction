#include "hip/hip_runtime.h"
#include "table.inc"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

struct vec3i {
  __host__ __device__ vec3i() {}
  __host__ __device__ vec3i(int x, int y, int z) : x(x), y(y), z(z) {}
  int x, y, z;
};

__device__ vec3i operator+(const vec3i &a, const vec3i &b) {
  return {a.x + b.x, a.y + b.y, a.z + b.z};
}
__device__ vec3i operator*(const vec3i &a, const int b) {
  return {a.x * b, a.y * b, a.z * b};
}
__device__ __host__ vec3i operator>>(const vec3i v, const int s) {
  return vec3i(v.x >> s, v.y >> s, v.z >> s);
}

__device__ __host__ long leftShift3(long x) {
  x = (x | x << 32) & 0x1f00000000ffffull;
  x = (x | x << 16) & 0x1f0000ff0000ffull;
  x = (x | x << 8) & 0x100f00f00f00f00full;
  x = (x | x << 4) & 0x10c30c30c30c30c3ull;
  x = (x | x << 2) & 0x1249249249249249ull;
  return x;
}

__device__ __host__ long mortonCode(const vec3i v) {
  return (leftShift3(uint32_t(v.z)) << 2) | (leftShift3(uint32_t(v.y)) << 1) |
         (leftShift3(uint32_t(v.x)) << 0);
}

__host__ __device__ bool operator==(const vec3i &a, const vec3i &b) {
  return a.x == b.x && a.y == b.y && a.z == b.z;
}

struct vec3f {
  __device__ vec3f() {}
  __device__ vec3f(const float x, const float y, const float z)
      : x(x), y(y), z(z) {}
  __host__ __device__ vec3f(const float f) : x(f), y(f), z(f) {}
  __host__ __device__ vec3f(const vec3i o) : x(o.x), y(o.y), z(o.z) {}

  float x, y, z;
};

__host__ __device__ vec3f operator+(const vec3f &a, const vec3f &b) {
  return {a.x + b.x, a.y + b.y, a.z + b.z};
}
__device__ vec3f operator-(const vec3f &a, const vec3f &b) {
  return {a.x - b.x, a.y - b.y, a.z - b.z};
}
__device__ vec3f operator*(const vec3f &a, const vec3f &b) {
  return {a.x * b.x, a.y * b.y, a.z * b.z};
}
__device__ vec3f operator*(const vec3f &a, const float b) {
  return {a.x * b, a.y * b, a.z * b};
}
__device__ bool operator==(const vec3f &a, const vec3f &b) {
  return a.x == b.x && a.y == b.y && a.z == b.z;
}
__host__ __device__ bool operator<(const vec3f &a, const vec3f &b) {
  return (a.x < b.x) ||
         ((a.x == b.x) && ((a.y < b.y) || (a.y == b.y) && (a.z < b.z)));
}
__device__ float4 operator+(const float4 &a, const float4 &b) {
  return {a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w};
}
__device__ float4 operator*(const float b, const float4 &a) {
  return {a.x * b, a.y * b, a.z * b, a.w * b};
}
__device__ bool operator==(const float4 &a, const float4 &b) {
  return a.x == b.x && a.y == b.y && a.z == b.z && a.w == b.w;
}

struct CellCoords {
  __device__ CellCoords neighbor(const vec3i &delta) const {
    return {lower + delta * (1 << level), level};
  }
  __device__ vec3f center() const {
    return vec3f(lower) + vec3f(0.5f * (1 << level));
  }
  vec3i lower;
  int level;
};

__host__ __device__ bool operator<(const CellCoords &a, const CellCoords &b) {
  return (a.lower < b.lower) || (a.lower == b.lower && b.level < b.level);
}

__host__ __device__ bool operator==(const CellCoords &a, const CellCoords &b) {
  return (a.lower == b.lower) && (a.level == b.level);
}

struct Cell : public CellCoords {
  __device__ float4 asDualVertex() const {
    return make_float4(center().x, center().y, center().z, scalar);
  }
  float scalar, field;
};

__host__ __device__ bool operator==(const Cell &a, const Cell &b) {
  return ((const CellCoords &)a == (const CellCoords &)b) &&
         (a.scalar == b.scalar);
}

__device__ bool operator!=(const Cell &a, const Cell &b) { return !(a == b); }

struct Morton {
  uint64_t morton;
  const Cell *cell;
};

struct CompareMorton0 {
  __device__ bool operator()(const Morton &a, const uint64_t b) {
    return a.morton < b;
  }
};

struct CompareMorton1 {
  __device__ bool operator()(const Morton &a, const Morton &b) {
    return a.morton < b.morton;
  }
};

struct TriangleVertex {
  vec3f position;
  uint32_t triangleAndVertexID;
};

struct CompareVertices {
  __device__ bool operator()(const TriangleVertex &lhs,
                             const TriangleVertex &rhs) const {
    const float4 a = (const float4 &)lhs;
    const float4 b = (const float4 &)rhs;

    return (const vec3f &)a < (const vec3f &)b;
  }
};

struct AMR {
  __device__ AMR(const Morton *const __restrict__ mortonArray,
                 const Cell *const __restrict__ cellArray, const int ncell,
                 const int maxlevel)
      : mortonArray(mortonArray), cellArray(cellArray), ncell(ncell),
        maxlevel(maxlevel) {}

  __device__ bool findActual(struct Cell &result, const CellCoords &coords) {
    const Morton *const __restrict__ begin = mortonArray;
    const Morton *const __restrict__ end = mortonArray + ncell;

    const Morton *it = thrust::system::detail::generic::scalar::lower_bound(
        begin, end, mortonCode(coords.lower), CompareMorton0());

    if (it == end)
      return false;

    const Cell found = *it->cell;
    if ((found.lower >> max(coords.level, found.level)) ==
        (coords.lower >> max(coords.level, found.level))
        // &&
        // (found.level >= coords.level)
    ) {
      result = found;
      return true;
    }

    if (it > begin) {
      const Cell found = *it[-1].cell;
      if ((found.lower >> max(coords.level, found.level)) ==
          (coords.lower >> max(coords.level, found.level))
          // &&
          // (found.level >= coords.level)
      ) {
        result = found;
        return true;
      }
    }

    return false;
  };

  const Cell *const __restrict__ cellArray;
  const int ncell;
  const int maxlevel;
  const Morton *const __restrict__ mortonArray;
};

__global__ void buildMortonArray(Morton *const __restrict__ mortonArray,
                                 const Cell *const __restrict__ cellArray,
                                 const int ncell) {
  const size_t tid = threadIdx.x + size_t(blockDim.x) * blockIdx.x;
  if (tid >= ncell)
    return;
  mortonArray[tid].morton = mortonCode(cellArray[tid].lower);
  mortonArray[tid].cell = &cellArray[tid];
}

struct IsoExtractor {
  __device__ IsoExtractor(const float isoValue, TriangleVertex *outputArray,
                          int outputArraySize, int *p_atomicCounter)
      : isoValue(isoValue), outputArray(outputArray),
        outputArraySize(outputArraySize), p_atomicCounter(p_atomicCounter) {}

  const float isoValue;
  TriangleVertex *const outputArray;
  const int outputArraySize;
  int *const p_atomicCounter;

  int __device__ allocTriangle() { return atomicAdd(p_atomicCounter, 1); }

  void __device__ doMarchingCubesOn(const vec3i mirror,
                                    const Cell zOrder[2][2][2]) {
    // we have OUR cells in z-order, but VTK case table assumes
    // everything is is VTK 'hexahedron' ordering, so let's rearrange
    // ... and while doing so, also make sure that we flip based on
    // which direction the parent cell created this dual from
    float4 vertex[8] = {
        zOrder[0 + mirror.z][0 + mirror.y][0 + mirror.x].asDualVertex(),
        zOrder[0 + mirror.z][0 + mirror.y][1 - mirror.x].asDualVertex(),
        zOrder[0 + mirror.z][1 - mirror.y][1 - mirror.x].asDualVertex(),
        zOrder[0 + mirror.z][1 - mirror.y][0 + mirror.x].asDualVertex(),
        zOrder[1 - mirror.z][0 + mirror.y][0 + mirror.x].asDualVertex(),
        zOrder[1 - mirror.z][0 + mirror.y][1 - mirror.x].asDualVertex(),
        zOrder[1 - mirror.z][1 - mirror.y][1 - mirror.x].asDualVertex(),
        zOrder[1 - mirror.z][1 - mirror.y][0 + mirror.x].asDualVertex()};

    int index = 0;
    for (int i = 0; i < 8; i++)
      if (vertex[i].w > isoValue)
        index += (1 << i);
    if (index == 0 || index == 0xff)
      return;

    for (const int8_t *edge = &vtkMarchingCubesTriangleCases[index][0];
         edge[0] > -1; edge += 3) {
      float4 triVertex[3];
      for (int ii = 0; ii < 3; ii++) {
        const int8_t *vert = vtkMarchingCubes_edges[edge[ii]];
        const float4 v0 = vertex[vert[0]];
        const float4 v1 = vertex[vert[1]];
        const float t = (isoValue - v0.w) / float(v1.w - v0.w);
        triVertex[ii] = (1.f - t) * v0 + t * v1;
      }

      if (triVertex[1] == triVertex[0])
        continue;
      if (triVertex[2] == triVertex[0])
        continue;
      if (triVertex[1] == triVertex[2])
        continue;

      const int triangleID = allocTriangle();
      if (triangleID >= 3 * outputArraySize)
        continue;

      for (int j = 0; j < 3; j++) {
        (int &)triVertex[j].w = (4 * triangleID + j);
        (float4 &)outputArray[3 * triangleID + j] = triVertex[j];
      }
    }
  }
};

__global__ void extractTriangles(const Morton *const __restrict__ mortonArray,
                                 const Cell *const __restrict__ cellArray,
                                 const int ncell, const int maxlevel,
                                 const float isoValue,
                                 TriangleVertex *__restrict__ outVertex,
                                 const int outVertexSize,
                                 int *p_numGeneratedTriangles) {
  AMR amr(mortonArray, cellArray, ncell, maxlevel);

  const size_t tid = threadIdx.x + size_t(blockDim.x) * blockIdx.x;

  const int workID = tid / 8;
  if (workID >= ncell)
    return;
  const int directionID = tid % 8;
  const Cell currentCell = cellArray[workID];

  const int dz = (directionID & 4) ? 1 : -1;
  const int dy = (directionID & 2) ? 1 : -1;
  const int dx = (directionID & 1) ? 1 : -1;

  Cell corner[2][2][2];
  for (int iz = 0; iz < 2; iz++)
    for (int iy = 0; iy < 2; iy++)
      for (int ix = 0; ix < 2; ix++) {
        const vec3i delta = vec3i(dx * ix, dy * iy, dz * iz);
        const CellCoords cornerCoords = currentCell.neighbor(delta);

        if (!amr.findActual(corner[iz][iy][ix], cornerCoords))
          // corner does not exist - currentcell is on a boundary, and
          // this is not a dual cell
          return;

        if (corner[iz][iy][ix].level < currentCell.level)
          // somebody else will generate this same cell from a finer
          // level...
          return;

        if (corner[iz][iy][ix].level == currentCell.level &&
            corner[iz][iy][ix] < currentCell)
          // this other cell will generate this dual cell...
          return;
      }

  IsoExtractor isoExtractor(isoValue, outVertex, outVertexSize,
                            p_numGeneratedTriangles);
  isoExtractor.doMarchingCubesOn({dx == -1, dy == -1, dz == -1}, corner);
}

__global__ void
createVertexArray(int *cnt, const TriangleVertex *const __restrict__ vertices,
                  int nvert, float3 *vert, int size, int3 *index) {
  int i, j, k, l, id, tid, *tri;
  TriangleVertex vertex;
  tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= nvert)
    return;
  vertex = vertices[tid];
  if (tid > 0 && vertex.position == vertices[tid - 1].position)
    return;
  id = atomicAdd(cnt, 1);
  if (id >= size)
    return;
  vert[id] = (float3 &)vertex.position;
  for (i = tid; i < nvert && vertices[i].position == vertex.position; i++) {
    j = vertices[i].triangleAndVertexID;
    k = j % 4;
    l = j / 4;
    tri = &index[l].x;
    tri[k] = id;
  }
}

static int comp(const void *av, const void *bv) {
  struct Cell *a, *b;
  a = (struct Cell *)av;
  b = (struct Cell *)bv;
  return mortonCode(a->lower) - mortonCode(b->lower);
}

int main(int argc, char **argv) {
  float isoValue, *attr;
  float3 *vert;
  int3 *tri;
  size_t numJobs;
  int Verbose, maxlevel, level, Found, blockSize, numBlocks;
  long i, j, nvert, ntri, ncell, size, nlost;
  FILE *file, *cell_file, *scalar_file, *field_file;
  int cell[4], ox, oy, oz;
  char attr_path[FILENAME_MAX], xyz_path[FILENAME_MAX], tri_path[FILENAME_MAX],
      xdmf_path[FILENAME_MAX], *attr_base, *xyz_base, *tri_base, *cell_path,
      *scalar_path, *field_path, *output_path, *end;
  struct Cell needl, *cells, *result;

  Verbose = 0;
  while (*++argv != NULL && argv[0][0] == '-')
    switch (argv[0][1]) {
    case 'h':
      fprintf(stderr,
              "Usage: iso [-v] in.cells in.scalar in.field isoValue mesh\n");
      exit(1);
    case 'v':
      Verbose = 1;
      break;
    case '-':
      argv++;
      goto positional;
    default:
      fprintf(stderr, "iso: error: unknown option '%s'\n", *argv);
      exit(1);
    }
positional:
  if ((cell_path = *argv++) == NULL) {
    fprintf(stderr, "iso: error: in.cells is not given\n");
    exit(1);
  }
  if ((scalar_path = *argv++) == NULL) {
    fprintf(stderr, "iso: error: in.scalar is not given\n");
    exit(1);
  }
  if ((field_path = *argv++) == NULL) {
    fprintf(stderr, "iso: error: in.field is not given\n");
    exit(1);
  }
  if (*argv == NULL) {
    fprintf(stderr, "iso: error: isoValue is no given\n");
    exit(1);
  }
  isoValue = strtod(*argv, &end);
  if (*end != '\0') {
    fprintf(stderr, "iso: error: '%s' is not a number\n", *argv);
    exit(1);
  }
  argv++;
  if ((output_path = *argv++) == NULL) {
    fprintf(stderr, "iso: error: out.mesh is not given\n");
    exit(1);
  }
  if ((cell_file = fopen(cell_path, "r")) == NULL) {
    fprintf(stderr, "iso: error: fail to open '%s'\n", cell_path);
    exit(1);
  }
  if ((scalar_file = fopen(scalar_path, "r")) == NULL) {
    fprintf(stderr, "iso: error: fail to open '%s'\n", scalar_path);
    exit(1);
  }
  if ((field_file = fopen(field_path, "r")) == NULL) {
    fprintf(stderr, "iso: error: fail to open '%s'\n", field_path);
    exit(1);
  }
  fseek(cell_file, 0, SEEK_END);
  size = ftell(cell_file);
  fseek(cell_file, 0, SEEK_SET);
  ncell = size / (4 * sizeof(int));
  if ((cells = (struct Cell *)malloc(ncell * sizeof *cells)) == NULL) {
    fprintf(stderr, "iso: error: malloc failed\n");
    exit(1);
  }
  ox = INT_MAX;
  oy = INT_MAX;
  oz = INT_MAX;
  maxlevel = 0;
  for (i = 0; i < ncell; i++) {
    if (fread(cell, sizeof(cell), 1, cell_file) != 1) {
      fprintf(stderr, "iso: error: fail to read '%s'\n", cell_path);
      exit(1);
    }
    cells[i].lower.x = cell[0];
    cells[i].lower.y = cell[1];
    cells[i].lower.z = cell[2];
    cells[i].level = cell[3];
    if (fread(&cells[i].scalar, sizeof(cells[i].scalar), 1, scalar_file) != 1) {
      fprintf(stderr, "iso: error: fail to read '%s'\n", scalar_path);
      exit(1);
    }
    if (fread(&cells[i].field, sizeof(cells[i].field), 1, field_file) != 1) {
      fprintf(stderr, "iso: error: fail to read '%s'\n", field_path);
      exit(1);
    }
    maxlevel = std::max(maxlevel, cells[i].level);
    ox = std::min(ox, cells[i].lower.x);
    oy = std::min(oy, cells[i].lower.y);
    oz = std::min(oz, cells[i].lower.z);
  }
  for (i = 0; i < ncell; i++) {
    cells[i].lower.x -= ox;
    cells[i].lower.y -= oy;
    cells[i].lower.z -= oz;
  }

  if (Verbose)
    fprintf(stderr, "iso: ncell, maxlevel, origin: %ld %d [%d %d %d]\n", ncell,
            maxlevel, ox, oy, oz);
  if (fclose(cell_file) != 0) {
    fprintf(stderr, "cylinder: error: fail to close '%s'\n", cell_path);
    exit(1);
  }
  if (fclose(scalar_file) != 0) {
    fprintf(stderr, "cylinder: error: fail to close '%s'\n", scalar_path);
    exit(1);
  }
  if (fclose(field_file) != 0) {
    fprintf(stderr, "cylinder: error: fail to close '%s'\n", field_path);
    exit(1);
  }
  qsort(cells, ncell, sizeof *cells, comp);
  thrust::device_vector<Cell> d_cells{cells, cells + ncell};
  thrust::device_vector<Morton> d_mortonArray(ncell);
  numJobs = ncell;
  blockSize = 512;
  numBlocks = (numJobs + blockSize - 1) / blockSize;
  buildMortonArray<<<numBlocks, blockSize>>>(
      thrust::raw_pointer_cast(d_mortonArray.data()),
      thrust::raw_pointer_cast(d_cells.data()), d_cells.size());
  hipDeviceSynchronize();
  thrust::sort(d_mortonArray.begin(), d_mortonArray.end(), CompareMorton1());

  hipDeviceSynchronize();
  thrust::device_vector<int> d_atomicCounter(1);
  thrust::device_vector<TriangleVertex> d_triangleVertices(0);
  d_atomicCounter[0] = 0;
  numJobs = 8 * ncell;
  blockSize = 512;
  numBlocks = (numJobs + blockSize - 1) / blockSize;
  extractTriangles<<<numBlocks, blockSize>>>(
      thrust::raw_pointer_cast(d_mortonArray.data()),
      thrust::raw_pointer_cast(d_cells.data()), d_cells.size(), maxlevel,
      isoValue, thrust::raw_pointer_cast(d_triangleVertices.data()),
      d_triangleVertices.size(),
      thrust::raw_pointer_cast(d_atomicCounter.data()));
  hipDeviceSynchronize();
  ntri = d_atomicCounter[0];
  d_triangleVertices.resize(3 * ntri);
  d_atomicCounter[0] = 0;
  numJobs = 8 * ncell;
  blockSize = 512;
  numBlocks = (numJobs + blockSize - 1) / blockSize;
  extractTriangles<<<numBlocks, blockSize>>>(
      thrust::raw_pointer_cast(d_mortonArray.data()),
      thrust::raw_pointer_cast(d_cells.data()), d_cells.size(), maxlevel,
      isoValue, thrust::raw_pointer_cast(d_triangleVertices.data()),
      d_triangleVertices.size(),
      thrust::raw_pointer_cast(d_atomicCounter.data()));
  hipDeviceSynchronize();
  thrust::sort(d_triangleVertices.begin(), d_triangleVertices.end(),
               CompareVertices());
  hipDeviceSynchronize();
  thrust::device_vector<float3> d_vert(0);
  thrust::device_vector<int3> d_tri(ntri);
  d_atomicCounter[0] = 0;
  numJobs = 3 * ntri;
  blockSize = 512;
  numBlocks = (numJobs + blockSize - 1) / blockSize;
  createVertexArray<<<numBlocks, blockSize>>>(
      thrust::raw_pointer_cast(d_atomicCounter.data()),
      thrust::raw_pointer_cast(d_triangleVertices.data()),
      d_triangleVertices.size(), thrust::raw_pointer_cast(d_vert.data()),
      d_vert.size(), thrust::raw_pointer_cast(d_tri.data()));
  hipDeviceSynchronize();
  nvert = d_atomicCounter[0];
  d_vert.resize(nvert);
  d_atomicCounter[0] = 0;
  numJobs = 3 * ntri;
  blockSize = 512;
  numBlocks = (numJobs + blockSize - 1) / blockSize;
  createVertexArray<<<numBlocks, blockSize>>>(
      thrust::raw_pointer_cast(d_atomicCounter.data()),
      thrust::raw_pointer_cast(d_triangleVertices.data()),
      d_triangleVertices.size(), thrust::raw_pointer_cast(d_vert.data()), nvert,
      thrust::raw_pointer_cast(d_tri.data()));
  hipDeviceSynchronize();
  assert(d_tri.size() == ntri);
  assert(d_vert.size() == nvert);
  if ((vert = (float3 *)malloc(nvert * sizeof *vert)) == NULL) {
    fprintf(stderr, "iso: error: malloc failed\n");
    exit(1);
  }
  if ((tri = (int3 *)malloc(ntri * sizeof *tri)) == NULL) {
    fprintf(stderr, "iso: error: malloc failed\n");
    exit(1);
  }
  thrust::copy(d_vert.begin(), d_vert.end(), vert);
  thrust::copy(d_tri.begin(), d_tri.end(), tri);

  snprintf(xyz_path, sizeof xyz_path, "%s.xyz.raw", output_path);
  snprintf(tri_path, sizeof tri_path, "%s.tri.raw", output_path);
  snprintf(attr_path, sizeof attr_path, "%s.attr.raw", output_path);
  snprintf(xdmf_path, sizeof xdmf_path, "%s.xdmf2", output_path);
  xyz_base = xyz_path;
  tri_base = tri_path;
  attr_base = attr_path;
  for (j = 0; xyz_path[j] != '\0'; j++) {
    if (xyz_path[j] == '/' && xyz_path[j + 1] != '\0') {
      xyz_base = &xyz_path[j + 1];
      tri_base = &tri_path[j + 1];
      attr_base = &attr_path[j + 1];
    }
  }
  if ((file = fopen(xyz_path, "w")) == NULL) {
    fprintf(stderr, "iso: error: fail to open '%s'\n", xyz_path);
    exit(1);
  }
  if (fwrite(vert, nvert * sizeof *vert, 1, file) != 1) {
    fprintf(stderr, "iso: error: fail to write '%s'\n", xyz_path);
    exit(1);
  }
  if (fclose(file) != 0) {
    fprintf(stderr, "iso: fail to close '%s'\n", xyz_path);
    exit(1);
  }
  if ((file = fopen(tri_path, "w")) == NULL) {
    fprintf(stderr, "iso: error: fail to open '%s'\n", tri_path);
    exit(1);
  }
  if (fwrite(tri, ntri * sizeof *tri, 1, file) != 1) {
    fprintf(stderr, "iso: error: fail to write '%s'\n", tri_path);
    exit(1);
  }
  if (fclose(file) != 0) {
    fprintf(stderr, "iso: fail to close '%s'\n", tri_path);
    exit(1);
  }

  if ((attr = (float *)malloc(nvert * sizeof *attr)) == NULL) {
    fprintf(stderr, "iso: error: malloc failed\n");
    exit(1);
  }
  nlost = 0;
  for (j = 0; j < nvert; j++) {
    Found = 0;
    needl.lower.x = vert[j].x;
    needl.lower.y = vert[j].y;
    needl.lower.z = vert[j].z;
    level = 0;
    for (;;) {
      result = (struct Cell *)bsearch(&needl, cells, ncell, sizeof(struct Cell),
                                      comp);
      if (result != NULL && level == result->level) {
        Found = 1;
        break;
      }
      if (level == maxlevel)
        break;
      level++;
      needl.lower.x &= (~0 << level);
      needl.lower.y &= (~0 << level);
      needl.lower.z &= (~0 << level);
    }
    if (Found) {
      attr[j] = result->field;
    } else {
      nlost++;
      attr[j] = 0;
    }
  }
  if (Verbose)
    fprintf(stderr, "iso: nlost/nvert: %ld/%ld\n", nlost, nvert);

  if ((file = fopen(attr_path, "w")) == NULL) {
    fprintf(stderr, "iso: error: fail to open '%s'\n", attr_path);
    exit(1);
  }
  if (fwrite(attr, nvert * sizeof *attr, 1, file) != 1) {
    fprintf(stderr, "iso: error: fail to write '%s'\n", attr_path);
    exit(1);
  }
  if (fclose(file) != 0) {
    fprintf(stderr, "iso: fail to close '%s'\n", attr_path);
    exit(1);
  }
  free(vert);
  free(tri);
  free(cells);
  free(attr);

  if ((file = fopen(xdmf_path, "w")) == NULL) {
    fprintf(stderr, "iso: error: fail to open '%s'\n", xdmf_path);
    exit(1);
  }
  fprintf(file,
          "<Xdmf\n"
          "    Version=\"2\">\n"
          "  <Domain>\n"
          "    <Grid>\n"
          "      <Topology\n"
          "         TopologyType=\"Triangle\"\n"
          "         Dimensions=\"%ld\">\n"
          "        <DataItem\n"
          "            Dimensions=\"%ld 3\"\n"
          "            NumberType=\"Int\"\n"
          "            Format=\"Binary\">\n"
          "          %s\n"
          "        </DataItem>\n"
          "      </Topology>\n"
          "      <Geometry>\n"
          "        <DataItem\n"
          "            Dimensions=\"%ld 3\"\n"
          "            Precision=\"4\"\n"
          "            Format=\"Binary\">\n"
          "          %s\n"
          "        </DataItem>\n"
          "      </Geometry>\n"
          "      <Attribute\n"
          "          Name=\"u\">\n"
          "        <DataItem\n"
          "            Dimensions=\"%ld\"\n"
          "            Precision=\"4\"\n"
          "            Format=\"Binary\">\n"
          "          %s\n"
          "        </DataItem>\n"
          "      </Attribute>\n"
          "    </Grid>\n"
          "  </Domain>\n"
          "</Xdmf>\n",
          ntri, ntri, tri_base, nvert, xyz_base, nvert, attr_base);
  if (fclose(file) != 0) {
    fprintf(stderr, "iso: fail to close '%s'\n", xdmf_path);
    exit(1);
  }
}
